
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void readPnm(char * fileName, 
		int &width, int &height, uchar3 * &pixels)
{
	FILE * f = fopen(fileName, "r");
	if (f == NULL)
	{
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	char type[3];
	fscanf(f, "%s", type);
	
	if (strcmp(type, "P3") != 0) // In this exercise, we don't touch other types
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	fscanf(f, "%i", &width);
	fscanf(f, "%i", &height);
	
	int max_val;
	fscanf(f, "%i", &max_val);
	if (max_val > 255) // In this exercise, we assume 1 byte per value
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	pixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	for (int i = 0; i < width * height; i++)
		fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

	fclose(f);
}

void writePnm(uchar3 * pixels, int width, int height, 
		char * fileName)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	

	fprintf(f, "P3\n%i\n%i\n255\n", width, height); 

	for (int i = 0; i < width * height; i++)
		fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);
	
	fclose(f);
}

__global__ void blurImgKernel(uchar3 * inPixels, int width, int height, 
		float * filter, int filterWidth, 
		uchar3 * outPixels)
{
	// TODO

	// Get data "row" and "column" indices
	int ix = threadIdx.x + blockIdx.x * blockDim.x;
	int iy = threadIdx.y + blockIdx.y * blockDim.y;

	// The center of the filter
	int centerIdx = filterWidth >> 1;
	
	// Convolution
	if (ix < width && iy < height)
	{
		// Temporary summation
		float sumX = 0.0f, sumY = 0.0f, sumZ = 0.0f;

		// Instead of indexing the filter from 0
		// We start from -centerIdx to +centerIdx
		// This helps computing data indices more naturally
		for (int rFilter = -centerIdx; rFilter < centerIdx + 1; rFilter++)
		{
			for (int cFilter = -centerIdx; cFilter < centerIdx + 1; cFilter++)
			{
				// The filter element to be used
				// Plus centerIdx to do the right 0-indexing
				int iFilter = (rFilter + centerIdx) * filterWidth + (cFilter + centerIdx);
				
				// rPatched and cPatched are pseudo indices
				// Assume that we are extending the original image "centerIdx" wider on each edge
				int rPatched = iy + rFilter, cPatched = ix + cFilter;

				// rConv and cConv are the actual indices to be used for the convolution
				// Here we make use of the nested ternary operator in C/C++
				// If (Patched is inside the original image) then Conv = Patched (internal)
				// Else If (Patched < 0) then Conv = 0 (lower bound)
				// Esle Conv = height/width - 1 (upper bound)
				int rConv = (rPatched >= 0 && rPatched <= height - 1) ? rPatched : (rPatched < 0) ? 0 : (height - 1);
				int cConv = (cPatched >= 0 && cPatched <= width - 1) ? cPatched : (cPatched < 0) ? 0 : (width - 1);
				
				// The image element to be multiplied at this step
				int iConv = rConv * width + cConv;

				// A single multiplication
				sumX += inPixels[iConv].x * filter[iFilter];
				sumY += inPixels[iConv].y * filter[iFilter];
				sumZ += inPixels[iConv].z * filter[iFilter];
			}
		}

		// The actual output element
		int i = iy * width + ix;	
		outPixels[i].x = sumX;
		outPixels[i].y = sumY;
		outPixels[i].z = sumZ;
	}
}

void blurImg(uchar3 * inPixels, int width, int height, float * filter, int filterWidth, 
		uchar3 * outPixels,
		bool useDevice=false, dim3 blockSize=dim3(1, 1))
{
	GpuTimer timer;
	timer.Start();
	if (useDevice == false)
	{
		// TODO
		// The center of the filter
		int centerIdx = filterWidth >> 1;
	
		for (int r = 0; r < height; r++)
		{
			for (int c = 0; c < height; c++)
			{
				// Convolution
				// Temporary summation
				float sumX = 0.0f, sumY = 0.0f, sumZ = 0.0f;

				// Instead of indexing the filter from 0
				// We start from -centerIdx to +centerIdx
				// This helps computing data indices more naturally
				for (int rFilter = -centerIdx; rFilter < centerIdx + 1; rFilter++)
				{
					for (int cFilter = -centerIdx; cFilter < centerIdx + 1; cFilter++)
					{
						// The filter element to be used
						// Plus centerIdx to do the right 0-indexing
						int iFilter = (rFilter + centerIdx) * filterWidth + (cFilter + centerIdx);
						
						// rPatched and cPatched are pseudo indices
						// Assume that we are extending the original image "centerIdx" wider on each edge
						int rPatched = r + rFilter, cPatched = c + cFilter;
						
						// rConv and cConv are the actual indices to be used for the convolution
						// Here we make use of the nested ternary operator in C/C++
						// If (Patched is inside the original image) then Conv = Patched (internal)
						// Else If (Patched < 0) then Conv = 0 (lower bound)
						// Esle Conv = height/width - 1 (upper bound)
						int rConv = (rPatched >= 0 && rPatched <= height - 1) ? rPatched : (rPatched < 0) ? 0 : (height - 1);
						int cConv = (cPatched >= 0 && cPatched <= width - 1) ? cPatched : (cPatched < 0) ? 0 : (width - 1);
						
						// The image element to be multiplied at this step
						int iConv = rConv * width + cConv;

						// A single multiplication
						sumX += inPixels[iConv].x * filter[iFilter];
						sumY += inPixels[iConv].y * filter[iFilter];
						sumZ += inPixels[iConv].z * filter[iFilter];
					}
				}

				// The actual output element
				int i = r * width + c;	
				outPixels[i].x = sumX;
				outPixels[i].y = sumY;
				outPixels[i].z = sumZ;
			}
		}
	}
	else // Use device
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, 0);
		printf("GPU name: %s\n", devProp.name);
		printf("GPU compute capability: %d.%d\n", devProp.major, devProp.minor);

		// TODO

		// Allocate device memories
		uchar3 *d_inPixels, *d_outPixels;
		float *d_filter;
		CHECK(hipMalloc(&d_inPixels, width * height * sizeof(uchar3)));
		CHECK(hipMalloc(&d_outPixels, width * height * sizeof(uchar3)));
		CHECK(hipMalloc(&d_filter, filterWidth * filterWidth * sizeof(float)))

		// Copy data to device memories
		CHECK(hipMemcpy(d_inPixels, inPixels, width * height * sizeof(uchar3), hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_filter, filter, filterWidth * filterWidth * sizeof(float), hipMemcpyHostToDevice));

		// Set grid size and call kernel (remember to check kernel error)
		dim3 gridSize((width - 1)/blockSize.x + 1, (height - 1)/blockSize.y + 1);
		blurImgKernel<<<gridSize, blockSize>>>(d_inPixels, width, height, d_filter, filterWidth, d_outPixels);
		hipError_t errorCode = hipGetLastError();
		if (hipSuccess != errorCode)
		{
			printf("Error: kernel did not run properly\n");
			printf("Code: %d, Reason: %s\n\n", errorCode, hipGetErrorString(errorCode));
			return;
		}

		// Copy result from device memories
		CHECK(hipMemcpy(outPixels, d_outPixels, width*height*sizeof(uchar3), hipMemcpyDeviceToHost));

		// Free device memories
		CHECK(hipFree(d_inPixels));
		CHECK(hipFree(d_outPixels));
		CHECK(hipFree(d_filter));
	}
	timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time (%s): %f ms\n", 
    		useDevice == true? "use device" : "use host", time);
}

float computeError(uchar3 * a1, uchar3 * a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)
	{
		err += abs((int)a1[i].x - (int)a2[i].x);
		err += abs((int)a1[i].y - (int)a2[i].y);
		err += abs((int)a1[i].z - (int)a2[i].z);
	}
	err /= (n * 3);
	return err;
}

char * concatStr(const char * s1, const char * s2)
{
    char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

int main(int argc, char ** argv)
{
	if (argc != 4 && argc != 6)
	{
		printf("The number of arguments is invalid\n");
		return EXIT_FAILURE;
	}

	// Read input image file
	int width, height;
	uchar3 * inPixels;
	readPnm(argv[1], width, height, inPixels);
	printf("Image size (width x height): %i x %i\n\n", width, height);

	// Read correct output image file
	int correctWidth, correctHeight;
	uchar3 * correctOutPixels;
	readPnm(argv[3], correctWidth, correctHeight, correctOutPixels);
	if (correctWidth != width || correctHeight != height)
	{
		printf("The shape of the correct output image is invalid\n");
		return EXIT_FAILURE;
	}

	// Set up a simple filter with blurring effect 
	int filterWidth = 9;
	float * filter = (float *)malloc(filterWidth * filterWidth * sizeof(float));
	for (int filterR = 0; filterR < filterWidth; filterR++)
	{
		for (int filterC = 0; filterC < filterWidth; filterC++)
		{
			filter[filterR * filterWidth + filterC] = 1. / (filterWidth * filterWidth);
		}
	}

	// Blur input image using host
	uchar3 * hostOutPixels = (uchar3 *)malloc(width * height * sizeof(uchar3)); 
	blurImg(inPixels, width, height, filter, filterWidth, hostOutPixels);
	
	// Compute mean absolute error between host result and correct result
	float hostErr = computeError(hostOutPixels, correctOutPixels, width * height);
	printf("Error: %f\n\n", hostErr);

	// Blur input image using device
	uchar3 * deviceOutPixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	dim3 blockSize(32, 32); // Default
	if (argc == 6)
	{
		blockSize.x = atoi(argv[4]);
		blockSize.y = atoi(argv[5]);
	}  
	blurImg(inPixels, width, height, filter, filterWidth, deviceOutPixels, true, blockSize);

	// Compute mean absolute error between device result and correct result
	float deviceErr = computeError(deviceOutPixels, correctOutPixels, width * height);
	printf("Error: %f\n\n", deviceErr);

	// Write results to files
	char * outFileNameBase = strtok(argv[2], "."); // Get rid of extension
	writePnm(hostOutPixels, width, height, concatStr(outFileNameBase, "_host.pnm"));
	writePnm(deviceOutPixels, width, height, concatStr(outFileNameBase, "_device.pnm"));

	// Free memories
	free(inPixels);
	free(correctOutPixels);
	free(hostOutPixels);
	free(deviceOutPixels);
	free(filter);
}
